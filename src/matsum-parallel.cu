
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

__global__ void matrix_sum(int *_C, int *_A, int *_B, int size) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < size)
    _C[i] = _A[i] + _B[i];
}

#define CUDACHECK(cmd) \
  do { \
    hipError_t e = cmd; \
    if ( e != hipSuccess) { \
      printf("Failed: Cuda error %s:%s '%s'\n", \
          __FILE__, __LINE__, \
          hipGetErrorString(e)); \
      exit(EXIT_FAILURE); \
    } \
  } while (0);


int main(int argc, char **argv) {
  //host variables
  int *A, 
      *B, 
      *C;
  int i, 
      j;
  double t;
  int size;
  int bytes;

  //device variables
  int *_A, 
      *_B, 
      *_C;

  // Input
  int rows, 
      cols;
  FILE *input;

  if (argc < 2) {
    fprintf(stderr, "Error: missing path to input file\n");
    return EXIT_FAILURE;
  }

  if ((input = fopen(argv[1], "r")) == NULL) {
    fprintf(stderr, "Error: could not open file\n");
    return EXIT_FAILURE;
  }

  fscanf(input, "%d", &rows);
  fscanf(input, "%d", &cols);

  size = rows * cols;
  bytes = sizeof(int) * size;

  // Allocate memory on the host
  A = (int *)malloc(bytes);
  B = (int *)malloc(bytes);
  C = (int *)malloc(bytes);

  // Allocate memory on the device
  CUDACHECK(hipMalloc(&_A, bytes));
  CUDACHECK(hipMalloc(&_B, bytes));
  CUDACHECK(hipMalloc(&_C, bytes));
  
  // Initialize memory
  for (i = 0; i < rows; i++) {
    for (j = 0; j < cols; j++) {
      A[i * cols + j] = B[i * cols + j] = i + j;
    }
  }

  // Copy data to device
  CUDACHECK(hipMemcpy(_A, A, bytes, hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(_B, B, bytes, hipMemcpyHostToDevice));

  // Compute matrix sum on device
  // Leave only the kernel and synchronize inside the timing region!
  int nBlocks = (size + 127)/128;
  t = omp_get_wtime();
  matrix_sum<<<nBlocks, 128>>>(_C, _A, _B, size);
  CUDACHECK(hipDeviceSynchronize());
  t = omp_get_wtime() - t;

  // Copy data back to host
  CUDACHECK(hipMemcpy(C, _C, bytes, hipMemcpyDeviceToHost));

  long long int sum = 0;

  // Keep this computation on the CPU
  for (i = 0; i < rows; i++) {
    for (j = 0; j < cols; j++) {
      sum += C[i * cols + j];
    }
  }

  fprintf(stdout, "%lli\n", sum);
  fprintf(stderr, "%lf\n", t);

  free(A);
  free(B);
  free(C);
  CUDACHECK(hipFree(_A));
  CUDACHECK(hipFree(_B));
  CUDACHECK(hipFree(_C));
}
